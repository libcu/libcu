#include "fsystem.h"
#include <sys/statcu.h>
#include <fcntlcu.h>
#include <sentinel-fcntlmsg.h>
#include <sentinel-unistdmsg.h>
#include <stdio.h> // panic's printf

__BEGIN_DECLS;

#ifdef __CUDA_ARCH__
__device__ int fcntl_(int fd, int cmd, ...) { va_list va; va_start(va, cmd); int r = vfcntl_(fd, cmd, va); va_end(va); return r; }
#endif
__device__ int vfcntl_(int fd, int cmd, va_list va) {
	if (ISHOSTHANDLE(fd)) { fcntl_fcntl msg(fd, cmd, va_arg(va, int), false); return msg.RC; }
	panic("Not Implemented");
	// (int fd, unsigned int cmd, unsigned long arg, struct file *filp)
	//	long err = -EINVAL;
	//	switch (cmd) {
	//	case F_DUPFD: err = f_dupfd(arg, filp, 0); break;
	//	case F_GETFD: err = get_close_on_exec(fd) ? FD_CLOEXEC : 0; break;
	//	case F_SETFD: err = 0; set_close_on_exec(fd, arg & FD_CLOEXEC); break;
	//	case F_GETFL: err = filp->f_flags; break;
	//	case F_SETFL: err = setfl(fd, filp, arg); break;
	//	case F_GETOWN: err = f_getown(filp); force_successful_syscall_return(); break;
	//	case F_SETOWN: f_setown(filp, arg, 1); err = 0; break;
	//	default:
	//		break;
	//	}
	//	return err;
	return 0;
}
#ifdef __USE_LARGEFILE64
#ifdef __CUDA_ARCH__
__device__ int fcntl64_(int fd, int cmd, ...) { va_list va; va_start(va, cmd); int r = vfcntl64_(fd, cmd, va); va_end(va); return r; }
#endif
__device__ int vfcntl64_(int fd, int cmd, va_list va) {
	if (ISHOSTHANDLE(fd)) { fcntl_fcntl msg(fd, cmd, va_arg(va, int), true); return msg.RC; }
	panic("Not Implemented");
	return 0;
}
#endif

#ifdef __CUDA_ARCH__
__device__ int open_(const char *file, int oflag, ...) { va_list va; va_start(va, oflag); int r = vopen_(file, oflag, va); va_end(va); return r; }
#endif
__device__ int vopen_(const char *file, int oflag, va_list va) {
	if (ISHOSTPATH(file)) { fcntl_open msg(file, oflag, va_arg(va, int), false); return msg.RC; }
	int fd; fsystemOpen(file, oflag, &fd); return fd;
}
#ifdef __USE_LARGEFILE64
#ifdef __CUDA_ARCH__
__device__ int open64_(const char *file, int oflag, ...) { va_list va; va_start(va, oflag); int r = vopen64_(file, oflag, va); va_end(va); return r; }
#endif
__device__ int vopen64_(const char *file, int oflag, va_list va) {
	if (ISHOSTPATH(file)) { fcntl_open msg(file, oflag, va_arg(va, int), true); return msg.RC; }
	int fd; fsystemOpen(file, oflag, &fd); return fd;
}
#endif

/* Close the file descriptor FD.  */
__device__ int close_(int fd) {
	if (ISHOSTHANDLE(fd)) { unistd_close msg(fd); return msg.RC; }
	fsystemClose(fd);
	return 0;
}

__END_DECLS;
