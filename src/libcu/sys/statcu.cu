#include <sys/statcu.h>
#include <sentinel-fcntlmsg.h>
#include "../fsystem.h"
#include <stdio.h> // panic's printf

/* Get file attributes for FILE and put them in BUF.  */
__device__ int stat_(const char *__restrict file, struct stat *__restrict buf, bool lstat) {
	if (ISHOSTPATH(file)) { fcntl_stat msg(file, buf, nullptr, false, lstat); return msg.RC; }
	panic("Not Implemented");
	return 0;
}

/* Get file attributes for the file, device, pipe, or socket that file descriptor FD is open on and put them in BUF.  */
__device__ int fstat_(int fd, struct stat *buf) {
	if (ISHOSTHANDLE(fd)) { fcntl_fstat msg(fd, buf, nullptr, false); return msg.RC; }
	panic("Not Implemented");
	return 0;
}

#ifdef __USE_LARGEFILE64
/* Get file attributes for FILE and put them in BUF.  */
__device__ int stat64_(const char *__restrict file, struct stat64 *__restrict buf, bool lstat) {
	if (ISHOSTPATH(file)) { fcntl_stat msg(file, nullptr, buf, lstat, true); return msg.RC; }
	panic("Not Implemented");
	return 0;
}

/* Get file attributes for the file, device, pipe, or socket that file descriptor FD is open on and put them in BUF.  */
__device__ int fstat64_(int fd, struct stat64 *buf) {
	if (ISHOSTHANDLE(fd)) { fcntl_fstat msg(fd, nullptr, buf, true); return msg.RC; }
	panic("Not Implemented");
	return 0;
}
#endif

/* Set file access permissions for FILE to MODE. If FILE is a symbolic link, this affects its target instead.  */
__device__ int chmod_(const char *file, mode_t mode) {
	if (ISHOSTPATH(file)) { fcntl_chmod msg(file, mode); return msg.RC; }
	panic("Not Implemented");
	return 0;
}

/* Set the file creation mask of the current process to MASK, and return the old creation mask.  */
__device__ mode_t umask_(mode_t mask) {
	panic("Not Implemented");
	return 0;
}

/* Create a new directory named PATH, with permission bits MODE.  */
__device__ int mkdir_(const char *path, mode_t mode) {
	if (ISHOSTPATH(path)) { fcntl_mkdir msg(path, mode); return msg.RC; }
	int r; fsystemMkdir(path, mode, &r); return r;
}

/* Create a new FIFO named PATH, with permission bits MODE.  */
__device__ int mkfifo_(const char *path, mode_t mode) {
	if (ISHOSTPATH(path)) { fcntl_mkfifo msg(path, mode); return msg.RC; }
	panic("Not Implemented");
	return 0;
}
