#include <stdiocu.h>
#include <setjmpcu.h>

__BEGIN_DECLS;

/* Store the calling environment in ENV, also saving the signal mask. Return 0.  */
__device__ int setjmp_(jmp_buf env) {
	panic("Not Implemented");
	return 0;
}

/* Jump to the environment saved in ENV, making the `setjmp' call there return VAL, or 1 if VAL is 0.  */
__device__ void longjmp_(jmp_buf env, int val) {
	panic("Not Implemented");
}

__END_DECLS;
