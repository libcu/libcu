#include "hip/hip_runtime.h"
#include <cuda_runtimecu.h>
#include <stdiocu.h>
#include <stringcu.h>
#include <assert.h>

static __global__ void g_memmove_speed();
static __global__ void g_strlen_speed();
static __global__ void g_strnlen_speed();
#define g_speed g_memmove_speed

int main() {
	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	// Launch test
	hipEventRecord(start);
	for (int i = 0; i < 1; i++)
		g_speed<<<1, 32>>>();
	hipEventRecord(stop);
	//
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Effective: %fn", milliseconds / 1e6);

Error:
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	// finish
	printf("\nPress any key to continue.\n");
	scanf("%c");

	return 0;
}

static __constant__ const char *_quickbrownfox =
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog.";

static __device__ char _buf0[] = "The quick brown fox jumped over the lazy dog.";
static __device__ char _buf1[50];

static __global__ void g_memmove_speed() {
	for (int i = 0; i < 1000; i++) {
		void *c = memmove(_buf1, nullptr, 0);
		assert(c == _buf1);
	}
	for (int i = 0; i < 1000; i++) {
		void *c = memmove(_buf1, _buf1, 10);
		assert(c == _buf1);
	}
	//for (int i = 0; i < 1000; i++) {
	//	void *c = memmove(_buf1, _buf0, 45);
	//}
}

static __global__ void g_strlen_speed() {
	for (int i = 0; i < 1000; i++) {
		int testLength = strlen(nullptr);
		assert(testLength == 0);
	}
	for (int i = 0; i < 1000; i++) {
		int testLength = strlen(_quickbrownfox);
		assert(testLength == 2196);
	}
}

static __global__ void g_strnlen_speed() {
	for (int i = 0; i < 1000; i++) {
		int testLength = strnlen(nullptr, 3000);
		assert(testLength == 0);
	}
	for (int i = 0; i < 1000; i++) {
		int testLength = strnlen(_quickbrownfox, 3000);
		assert(testLength == 2196);
	}
}