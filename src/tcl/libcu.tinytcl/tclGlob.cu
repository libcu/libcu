#include "hip/hip_runtime.h"
// tclGlob.c --
//
//	This file provides procedures and commands for file name manipulation, such as tilde expansion and globbing.
//
// Copyright 1990-1991 Regents of the University of California
// Permission to use, copy, modify, and distribute this software and its documentation for any purpose and without
// fee is hereby granted, provided that the above copyright notice appear in all copies.  The University of California
// makes no representations about the suitability of this software for any purpose.  It is provided "as is" without
// express or implied warranty.

#include "tclInt.h"
#include "tclGpu.h"

// The structure below is used to keep track of a globbing result being built up (i.e. a partial list of file names).  The list grows dynamically to be as big as needed.
typedef struct {
	char *result;		// Pointer to result area.
	int totalSpace;		// Total number of characters allocated for result.
	int spaceUsed;		// Number of characters currently in use to hold the partial result (not including the terminating NULL). */
	int dynamic;		// 0 means result is static space, 1 means it's dynamic.
} GlobResult;

// Declarations for procedures local to this file:
static __device__ void AppendResult(Tcl_Interp *interp, char *dir, char *separator, char *name, int nameLength);
static __device__ int DoGlob(Tcl_Interp *interp, char *dir, char *rem);

/*
*----------------------------------------------------------------------
*
* AppendResult --
*	Given two parts of a file name (directory and element within directory), concatenate the two together and append them to
*	the result building up in interp.
*
* Results:
*	There is no return value.
*
* Side effects:
*	Interp->result gets extended.
*
*----------------------------------------------------------------------
*/
static __device__ void AppendResult(Tcl_Interp *interp, char *dir, char *separator, char *name, int nameLength)
{
	// Next, see if we can put together a valid list element from dir and name by calling Tcl_AppendResult.
	int dirFlags;
	if (*dir == 0) {
		dirFlags = 0;
	} else {
		Tcl_ScanElement(dir, &dirFlags);
	}
	char saved = name[nameLength];
	name[nameLength] = 0;
	int nameFlags;
	Tcl_ScanElement(name, &nameFlags);
	if (!dirFlags && !nameFlags) {
		if (*interp->result != 0) {
			Tcl_AppendResult(interp, " ", dir, separator, name, (char *)NULL);
		} else {
			Tcl_AppendResult(interp, dir, separator, name, (char *)NULL);
		}
		name[nameLength] = saved;
		return;
	}

	// This name has weird characters in it, so we have to convert it to a list element.  To do that, we have to merge the characters
	// into a single name.  To do that, malloc a buffer to hold everything.
	char *p = (char *)_allocFast((unsigned)(strlen(dir) + strlen(separator) + nameLength + 1));
	sprintf(p, "%s%s%s", dir, separator, name);
	name[nameLength] = saved;
	Tcl_AppendElement(interp, p, 0);
	_freeFast(p);
}

/*
*----------------------------------------------------------------------
*
* DoGlob --
*	This recursive procedure forms the heart of the globbing code.  It performs a depth-first traversal of the tree
*	given by the path name to be globbed.
*
* Results:
*	The return value is a standard Tcl result indicating whether an error occurred in globbing.  After a normal return the
*	result in interp will be set to hold all of the file names given by the dir and rem arguments.  After an error the
*	result in interp will hold an error message.
*
* Side effects:
*	None.
*
*----------------------------------------------------------------------
*/
#undef STATIC_SIZE
static __device__ int DoGlob(Tcl_Interp *interp, char *dir, char *rem)
{
#define STATIC_SIZE 200 // When generating information for the next lower call, use static areas if the name is short, and malloc if the name is longer.
	// When this procedure is entered, the name to be globbed may already have been partly expanded by ancestor invocations of
	// DoGlob.  The part that's already been expanded is in "dir" (this may initially be empty), and the part still to expand
	// is in "rem".  This procedure expands "rem" one level, making recursive calls to itself if there's still more stuff left
	// in the remainder.
	register char *p;

	// Figure out whether we'll need to add a slash between the directory name and file names within the directory when concatenating them together.
	char *separator;
	if (dir[0] == 0 || (dir[0] == '/' && dir[1] == 0) ? "" : "/") {
		separator = "";
	} else {
		separator = "/";
	}

	// First, find the end of the next element in rem, checking along the way for special globbing characters.
	bool gotSpecial = false;
	char *openBrace = NULL, *closeBrace = NULL;
	for (p = rem; ; p++) {
		register char c = *p;
		if (c == '\0' || c == '/') {
			break;
		}
		if (c == '{' && !openBrace) {
			openBrace = p;
		}
		if (c == '}' && !closeBrace) {
			closeBrace = p;
		}
		if (c == '*' || c == '[' || c == '\\' || c == '?') {
			gotSpecial = true;
		}
	}

	// If there is an open brace in the argument, then make a recursive call for each element between the braces.  In this case, the
	// recursive call to DoGlob uses the same "dir" that we got. If there are several brace-pairs in a single name, we just handle
	// one here, and the others will be handled in recursive calls.
	if (openBrace) {
		char static1[STATIC_SIZE];
		if (!closeBrace) {
			Tcl_ResetResult(interp);
			interp->result = "unmatched open-brace in file name";
			return TCL_ERROR;
		}
		int remLength = strlen(rem) + 1;
		char *newRem;
		if (remLength <= STATIC_SIZE) {
			newRem = static1;
		} else {
			newRem = (char *)_allocFast((unsigned)remLength);
		}
		int l1 = (int)(openBrace-rem);
		strncpy(newRem, rem, l1);
		for (p = openBrace; *p != '}'; ) {
			char *element = p+1;
			for (p = element; ((*p != '}') && (*p != ',')); p++) { } /* Empty loop body:  just find end of this element. */
			int l2 = (int)(p - element);
			strncpy(newRem+l1, element, l2);
			strcpy(newRem+l1+l2, closeBrace+1);
			if (DoGlob(interp, dir, newRem) != TCL_OK) {
				return TCL_ERROR;
			}
		}
		if (remLength > STATIC_SIZE) {
			_freeFast(newRem);
		}
		return TCL_OK;
	}

	// If there were any pattern-matching characters, then scan through the directory to find all the matching names.
	int result;
	if (gotSpecial) {
		// Be careful not to do any actual file system operations on a directory named "";  instead, use ".".  This is needed because
		// some versions of UNIX don't treat "" like "." automatically.
		char *dirName;
		if (*dir == '\0') {
			dirName = ".";
		} else {
			dirName = dir;
		}
		struct stat statBuf;
		if (stat(dirName, &statBuf) != 0 || !S_ISDIR(statBuf.st_mode)) {
			return TCL_OK;
		}
		DIR *d = opendir(dirName);
		if (d == NULL) {
			Tcl_ResetResult(interp);
			Tcl_AppendResult(interp, "couldn't read directory \"", dirName, "\": ", Tcl_OSError(interp), (char *)NULL);
			return TCL_ERROR;
		}
		int l1 = strlen(dir);
		int l2 = (int)(p - rem);
		char static2[STATIC_SIZE];
		char *pattern;
		if (l2 < STATIC_SIZE) {
			pattern = static2;
		} else {
			pattern = (char *)_allocFast((unsigned)(l2+1));
		}
		strncpy(pattern, rem, l2);
		pattern[l2] = '\0';
		result = TCL_OK;
		while (true) {
			struct dirent *entryPtr = readdir(d);
			if (entryPtr == NULL) {
				break;
			}
			// Don't match names starting with "." unless the "." is present in the pattern.
			if (*entryPtr->d_name == '.' && *pattern != '.') {
				continue;
			}
			if (Tcl_StringMatch(entryPtr->d_name, pattern)) {
				int nameLength = strlen(entryPtr->d_name);
				if (*p == 0) {
					AppendResult(interp, dir, separator, entryPtr->d_name, nameLength);
				} else {
					char static1[STATIC_SIZE];
					char *newDir;
					if ((l1+nameLength+2) <= STATIC_SIZE) {
						newDir = static1;
					} else {
						newDir = (char *)_allocFast((unsigned)(l1+nameLength+2));
					}
					sprintf(newDir, "%s%s%s", dir, separator, entryPtr->d_name);
					result = DoGlob(interp, newDir, p+1);
					if (newDir != static1) {
						_freeFast(newDir);
					}
					if (result != TCL_OK) {
						break;
					}
				}
			}
		}
		closedir(d);
		if (pattern != static2) {
			_freeFast(pattern);
		}
		return result;
	}

	// This is the simplest case:  just another path element.  Move it to the dir side and recurse (or just add the name to the
	// list, if we're at the end of the path).
	if (*p == 0) {
		AppendResult(interp, dir, separator, rem, (int)(p-rem));
	} else {
		int l1 = strlen(dir);
		int l2 = l1 + (int)(p - rem) + 2;
		char static1[STATIC_SIZE];
		char *newDir;
		if (l2 <= STATIC_SIZE) {
			newDir = static1;
		} else {
			newDir = (char *) _allocFast((unsigned) l2);
		}
		sprintf(newDir, "%s%s%.*s", dir, separator, (int)(p-rem), rem);
		result = DoGlob(interp, newDir, p+1);
		if (newDir != static1) {
			_freeFast(newDir);
		}
		if (result != TCL_OK) {
			return TCL_ERROR;
		}
	}
	return TCL_OK;
}

/*
*----------------------------------------------------------------------
*
* Tcl_TildeSubst --
*	Given a name starting with a tilde, produce a name where the tilde and following characters have been replaced by
*	the home directory location for the named user.
*
* Results:
*	The result is a pointer to a static string containing the new name.  This name will only persist until the next
*	call to Tcl_TildeSubst;  save it if you care about it for the long term.  If there was an error in processing the
*	tilde, then an error message is left in interp->result and the return value is NULL.
*
* Side effects:
*	None that the caller needs to worry about.
*
*----------------------------------------------------------------------
*/
#if TCL_GETWD
__device__ char *Tcl_TildeSubst(Tcl_Interp *interp, char *name)
{
#define STATIC_BUF_SIZE 50
	static char staticBuf[STATIC_BUF_SIZE];
	static int curSize = STATIC_BUF_SIZE;
	static char *curBuf = staticBuf;
	int length;
	register char *p;

	if (name[0] != '~') {
		return name;
	}

	// First, find the directory name corresponding to the tilde entry.
	bool fromPw = false;
	char *dir;
	if (name[1] == '/' || name[1] == '\0') {
		dir = getenv("HOME");
		if (dir == NULL) {
			Tcl_ResetResult(interp);
			Tcl_AppendResult(interp, "couldn't find HOME environment ", "variable to expand \"", name, "\"", (char *)NULL);
			return NULL;
		}
		p = name+1;
	} else {
		struct passwd *pwPtr;
		for (p = &name[1]; *p != 0 && *p != '/'; p++) { } // Null body;  just find end of name.
		length = p-&name[1];
		if (length >= curSize) {
			length = curSize-1;
		}
		memcpy(curBuf, (name+1), length);
		curBuf[length] = '\0';
		pwPtr = getpwnam(curBuf);
		if (pwPtr == NULL) {
			endpwent();
			Tcl_ResetResult(interp);
			Tcl_AppendResult(interp, "user \"", curBuf, "\" doesn't exist", (char *)NULL);
			return NULL;
		}
		dir = pwPtr->pw_dir;
		fromPw = true;
	}

	// Grow the buffer if necessary to make enough space for the full file name.
	length = strlen(dir) + strlen(p);
	if (length >= curSize) {
		if (curBuf != staticBuf) {
			_freeFast(curBuf);
		}
		curSize = length + 1;
		curBuf = (char *)_allocFast((unsigned)curSize);
	}

	// Finally, concatenate the directory name with the remainder of the path in the buffer.
	strcpy(curBuf, dir);
	strcat(curBuf, p);
	if (fromPw) {
		endpwent();
	}
	return curBuf;
}
#else
__device__ char *Tcl_TildeSubst(Tcl_Interp *interp, char *name)
{
	return name;
}
#endif

/*
*----------------------------------------------------------------------
*
* Tcl_GlobCmd --
*	This procedure is invoked to process the "glob" Tcl command. See the user documentation for details on what it does.
*
* Results:
*	A standard Tcl result.
*
* Side effects:
*	See the user documentation.
*
*----------------------------------------------------------------------
*/
__device__ int Tcl_GlobCmd(ClientData dummy, Tcl_Interp *interp, int argc, const char *args[])
{
	if (argc < 2) {
notEnoughArgs:
		Tcl_AppendResult(interp, "wrong # args: should be \"", args[0], " ?-nocomplain? name ?name ...?\"", (char *)NULL);
		return TCL_ERROR;
	}
	bool noComplain = false;
	if (args[1][0] == '-' && !strcmp(args[1], "-nocomplain")) {
		if (argc < 3) {
			goto notEnoughArgs;
		}
		noComplain = true;
	}

	int i;
	for (i = 1 + noComplain; i < argc; i++) {
		// Do special checks for names starting at the root and for names beginning with ~.  Then let DoGlob do the rest.
		char *thisName = (char *)args[i];
#if TCL_GETWD
		if (*thisName == '~') {
			thisName = Tcl_TildeSubst(interp, thisName);
			if (thisName == NULL) {
				return TCL_ERROR;
			}
		}
#endif
		int result;
		if (*thisName == '/') {
			result = DoGlob(interp, "/", thisName+1);
		} else {
			result = DoGlob(interp, "", thisName);
		}
		if (result != TCL_OK) {
			return result;
		}
	}
	if (!*interp->result && !noComplain) {
		char *sep = "";
		Tcl_AppendResult(interp, "no files matched glob pattern", (argc == 2 ? " \"" : "s \""), (char *)NULL);
		for (i = 1; i < argc; i++) {
			Tcl_AppendResult(interp, sep, args[i], (char *)NULL);
			sep = " ";
		}
		Tcl_AppendResult(interp, "\"", (char *)NULL);
		return TCL_ERROR;
	}
	return TCL_OK;
}
