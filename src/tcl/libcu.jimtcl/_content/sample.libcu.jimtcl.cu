#include "hip/hip_runtime.h"
#include <cuda_runtimecu.h>

void addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    addWithCuda(c, a, b, arraySize);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaErrorCheck(hipDeviceReset());

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
	cudaDeviceHeap deviceHeap;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaErrorCheckF(hipSetDevice(0), goto Error);

	deviceHeap = cudaDeviceHeapCreate();

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaErrorCheckF(hipMalloc((void**)&dev_c, size * sizeof(int)), goto Error);
    cudaErrorCheckF(hipMalloc((void**)&dev_a, size * sizeof(int)), goto Error);
    cudaErrorCheckF(hipMalloc((void**)&dev_b, size * sizeof(int)), goto Error);

    // Copy input vectors from host memory to GPU buffers.
    cudaErrorCheckF(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice), goto Error);
    cudaErrorCheckF(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice), goto Error);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
	
    // Check for any errors launching the kernel
    cudaErrorCheckF(hipGetLastError(), goto Error);
    
	// cudaDeviceHeapSynchronize..
	cudaErrorCheckF(cudaDeviceHeapSynchronize(deviceHeap), goto Error);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaErrorCheckF(hipDeviceSynchronize(), goto Error);

    // Copy output vector from GPU buffer to host memory.
    cudaErrorCheckF(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost), goto Error);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
	cudaDeviceHeapDestroy(deviceHeap);
}
