#include "hip/hip_runtime.h"
#pragma region License
/*
* jim-clock.c
*
* Implements the clock command
*/
#pragma endregion

/* For strptime() */
#ifndef _XOPEN_SOURCE
#define _XOPEN_SOURCE 500
#endif

//#include <stdlib.h>
//#include <string.h>
//#include <stdio.h>
#include <timecu.h>
#include "jimautoconf.h"
#include "jim-subcmd.h"
#ifdef HAVE_SYS_TIME_H
#include <sys/time.h>
#endif

static __device__ int clock_cmd_format(Jim_Interp *interp, int argc, Jim_Obj *const *argv)
{
	// How big is big enough?
	if (argc == 2 || (argc == 3 && !Jim_CompareStringImmediate(interp, argv[1], "-format")))
		return -1;
	const char *format = (argc == 3 ? Jim_String(argv[2]) : "%a %b %d %H:%M:%S %Z %Y");
	long seconds;
	if (Jim_GetLong(interp, argv[0], &seconds) != JIM_OK)
		return JIM_ERROR;
	time_t t = seconds;
	char buf[100];
	if (!strftime(buf, sizeof(buf), format, localtime(&t))) {
		Jim_SetResultString(interp, "format string too long", -1);
		return JIM_ERROR;
	}
	Jim_SetResultString(interp, buf, -1);
	return JIM_OK;
}

#ifdef HAVE_STRPTIME
static __device__ int clock_cmd_scan(ClientData dummy, Jim_Interp *interp, int argc, Jim_Obj *const *argv)
{
	if (!Jim_CompareStringImmediate(interp, argv[1], "-format"))
		return -1;
	// Initialise with the current date/time
	struct tm tm;
	time_t now = time(0);
	localtime_r(&now, &tm);
	char *pt = strptime(Jim_String(argv[0]), Jim_String(argv[2]), &tm);
	if (pt == 0 || *pt != 0) {
		Jim_SetResultString(interp, "Failed to parse time according to format", -1);
		return JIM_ERROR;
	}
	// Now convert into a time_t
	Jim_SetResultInt(interp, mktime(&tm));
	return JIM_OK;
}
#endif

static __device__ int clock_cmd_seconds(Jim_Interp *interp, int argc, Jim_Obj *const *argv)
{
	Jim_SetResultInt(interp, time(NULL));
	return JIM_OK;
}

static __device__ int clock_cmd_micros(Jim_Interp *interp, int argc, Jim_Obj *const *argv)
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	Jim_SetResultInt(interp, (jim_wide)tv.tv_sec * 1000000 + tv.tv_usec);
	return JIM_OK;
}

static __device__ int clock_cmd_millis(Jim_Interp *interp, int argc, Jim_Obj *const *argv)
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	Jim_SetResultInt(interp, (jim_wide)tv.tv_sec * 1000 + tv.tv_usec / 1000);
	return JIM_OK;
}

__constant__ static const jim_subcmd_type clock_command_table[] = {
	{ "seconds", NULL, clock_cmd_seconds, 0, 0, }, /* Description: Returns the current time as seconds since the epoch */
	{ "clicks", NULL, clock_cmd_micros, 0, 0, }, /* Description: Returns the current time in 'clicks' */
	{ "microseconds", NULL, clock_cmd_micros, 0, 0, }, /* Description: Returns the current time in microseconds */
	{ "milliseconds", NULL, clock_cmd_millis, 0, 0, }, /* Description: Returns the current time in milliseconds */
	{ "format", "seconds ?-format format?", clock_cmd_format, 1, 3, }, /* Description: Format the given time */
#ifdef HAVE_STRPTIME
	{ "scan", "str -format format", clock_cmd_scan, 3, 3, }, /* Description: Determine the time according to the given format */
#endif
	{ NULL }
};

__device__ int Jim_clockInit(Jim_Interp *interp)
{
	if (Jim_PackageProvide(interp, "clock", "1.0", JIM_ERRMSG))
		return JIM_ERROR;
	Jim_CreateCommand(interp, "clock", Jim_SubCmdProc, (void *)clock_command_table, NULL);
	return JIM_OK;
}
