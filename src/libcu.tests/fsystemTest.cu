#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <stringcu.h>
#include "../libcu/fsystem.h"
#include <assert.h>

static __global__ void g_fsystem_test1() {
	printf("fsystem_test1\n");
	char newPath[MAX_PATH];
	// ABSOLUTE
	strcpy(__cwd, ":\\test");
	expandPath(":\\", newPath); int a0a = !strcmp(newPath, ":");
	expandPath(":/one", newPath); int a1a = !strcmp(newPath, ":\\one");
	expandPath(":\\one", newPath); int a2a = !strcmp(newPath, ":\\one");
	expandPath(":\\one\\", newPath); int a3a = !strcmp(newPath, ":\\one");
	assert(a0a && a1a && a2a && a3a);

	expandPath(":\\.", newPath); int b0a = !strcmp(newPath, ":");
	expandPath(":\\one\\.", newPath); int b1a = !strcmp(newPath, ":\\one");
	expandPath(":\\one\\.\\", newPath); int b2a = !strcmp(newPath, ":\\one");
	expandPath(":\\one\\.\\two", newPath); int b3a = !strcmp(newPath, ":\\one\\two");
	assert(b0a && b1a && b2a && b3a);

	expandPath(":\\one\\..\\two", newPath); int c0a = !strcmp(newPath, ":\\two");
	expandPath(":\\one\\..\\two\\three", newPath); int c1a = !strcmp(newPath, ":\\two\\three");
	assert(c0a && c1a);

	// ROOT
	strcpy(__cwd, ":\\test");
	expandPath("\\.", newPath); int d0a = !strcmp(newPath, ":");
	expandPath("\\one", newPath); int d1a = !strcmp(newPath, ":\\one");
	assert(d0a && d1a);

	// RELATIVE
	strcpy(__cwd, ":\\test");
	expandPath(".", newPath); int e0a = !strcmp(newPath, ":\\test"); //printf("%s\n", newPath);
	expandPath("one", newPath); int e1a = !strcmp(newPath, ":\\test\\one"); //printf("%s\n", newPath);
	//assert(e0a && e1a);

	// CHDIR
	strcpy(__cwd, ":\\test");
	int f0a = fsystemChdir(":\\"); int f0b = !strcmp(__cwd, ":\\");
	//assert(f0a);

	// OPENDIR
	dirEnt_t *g0a = fsystemOpendir(":\\"); int g0b = !strcmp(__cwd, ":\\");
	//assert(g0a);

	// RENAME
	int h0a = fsystemRename(":\\", ":\\"); int h0b = !strcmp(__cwd, ":\\");
	//assert(h0a);

	// UNLINK
	int i0a = fsystemUnlink(":\\", false); int i0b = !strcmp(__cwd, ":\\");
	//assert(i0a);

	// MKDIR
	int r;
	dirEnt_t *j0a = fsystemMkdir(":\\", 0, &r); int j0b = !strcmp(__cwd, ":\\");
	//assert(j0a);

	// OPEN
	int fd;
	dirEnt_t *k0a = fsystemOpen(":\\", 0, &fd); int k0b = !strcmp(__cwd, ":\\");
	//assert(k0a);

	// RESET
	fsystemReset();
}
hipError_t fsystem_test1() { g_fsystem_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }
