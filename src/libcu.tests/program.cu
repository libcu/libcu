#include <cuda_runtimecu.h>
#include <sentinel.h>
#include <stdlibcu.h>
#include <stdiocu.h>

hipError_t crtdefs_test1();
hipError_t ctype_test1();
hipError_t dirent_test1();
hipError_t errno_test1();
hipError_t fcntl_test1();
hipError_t fsystem_test1();
hipError_t grp_test1();
hipError_t pwd_test1();
hipError_t regex_test1();
hipError_t sentinel_test1();
hipError_t setjmp_test1();
hipError_t stddef_test1();
hipError_t stdio_test1();
hipError_t stdio_64bit();
hipError_t stdio_ganging();
hipError_t stdio_scanf();
hipError_t stdlib_test1();
hipError_t stdlib_strtol();
hipError_t stdlib_strtoq();
hipError_t string_test1();
hipError_t time_test1();
hipError_t unistd_test1();

#if _HASPAUSE
#define mainPause(fmt) { printf(fmt"\n"); char c; scanf("%c", &c); }
#else
#define mainPause(fmt) { printf(fmt"\n"); }
#endif

int main(int argc, char **argv) {
	int testId = argv[1] ? atoi(argv[1]) : 16;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(gpuGetMaxGflopsDevice());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024 * 5));
	sentinelServerInitialize();
	sentinelRegisterFileUtils();

	// Launch test
	switch (testId) {
	case 0: mainPause("Press any key to continue."); break;
	case 1: cudaStatus = crtdefs_test1(); break;
	case 2: cudaStatus = ctype_test1(); break;
	case 3: cudaStatus = dirent_test1(); break;
	case 4: cudaStatus = errno_test1(); break;
	case 5: cudaStatus = fcntl_test1(); break;
	case 6: cudaStatus = grp_test1(); break;
	case 7: cudaStatus = pwd_test1(); break;
	case 8: cudaStatus = regex_test1(); break;
	case 9: cudaStatus = sentinel_test1(); break;
	case 10: cudaStatus = setjmp_test1(); break;
	case 11: cudaStatus = stddef_test1(); break;
	case 12: cudaStatus = stdio_test1(); break; // assert
	case 13: cudaStatus = stdio_64bit(); break;
	case 14: cudaStatus = stdio_ganging(); break;
	case 15: cudaStatus = stdio_scanf(); break;
	case 16: cudaStatus = stdlib_test1(); break; // assert
	case 17: cudaStatus = stdlib_strtol(); break;
	case 18: cudaStatus = stdlib_strtoq(); break;
	case 19: cudaStatus = string_test1(); break;
	case 20: cudaStatus = time_test1(); break;
	case 21: cudaStatus = unistd_test1(); break; // missing device, throws on fast run
		// default
	default: cudaStatus = crtdefs_test1(); break;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// finish
	mainPause("SUCCESS");

Error:
	sentinelServerShutdown();

	// close
	if (cudaStatus != hipSuccess) {
		// finish
		mainPause("ERROR");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}
