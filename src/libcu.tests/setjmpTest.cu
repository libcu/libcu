#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <setjmpcu.h>
#include <assert.h>

static __global__ void g_setjmp_test1() {
	printf("setjmp_test1\n");

	//// SETJMP ////
	//extern __device__ int setjmp_(jmp_buf env);
	// a0a = setjmp - not:implemented

	//// __SIGSETJMP, _SETJMP ////
	////nosupport: extern int __sigsetjmp_(struct __jmp_buf_tag env[1], int savemask);
	////nosupport: extern int _setjmp_(struct __jmp_buf_tag env[1]);

	//// LONGJMP ////
	//extern __device__ void longjmp_(jmp_buf env, int val);
	// b0a = longjmp - not:implemented

}
hipError_t setjmp_test1() { g_setjmp_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }
