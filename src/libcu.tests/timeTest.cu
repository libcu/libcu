#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <timecu.h>
#include <assert.h>

static __global__ void g_time_test1() {
	printf("time_test1\n");
	time_t rawtime; time(&rawtime);
	struct tm *timeinfo = localtime(&rawtime);

	//// CLOCK, TIME, DIFFTIME, MKTIME ////
	////builtin: extern __device__ clock_t clock();
	//extern __device__ time_t time_(time_t *timer);
	//extern __device__ double difftime_(time_t time1, time_t time0);
	//extern __device__ time_t mktime_(struct tm *tp); #sentinel
	time_t a0a = time(nullptr); time_t a0b; time_t a0c = time(&a0b); assert(a0a != -1 && a0b != -1 && a0c != -1);
	double a1a = difftime(1, 2); assert(a1a == -1);

	//// STRFTIME ////
	//extern size_t strftime_(char *__restrict s, size_t maxsize, const char *__restrict format, const struct tm *__restrict tp); #sentinel
	char b0_buf[80];
	int b0a = strftime(b0_buf, sizeof(b0_buf), "Now it's %I:%M%p.", timeinfo);
	assert(!strncmp(b0_buf, "Now it's ", 9));

	//// GMTIME ////
	//extern __device__ struct tm *gmtime_(const time_t *timer);
	struct tm *c0a = gmtime(&rawtime); assert(c0a);

	//// ASCTIME, CTIME ////
	//extern __device__ char *asctime_(const struct tm *tp);
	//__forceinline__ __device__ char *ctime_(const time_t *timer);
	char *d0a = asctime(timeinfo); assert(d0a);
	char *d1a = ctime(&rawtime); assert(d1a);
}
hipError_t time_test1() { g_time_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }
