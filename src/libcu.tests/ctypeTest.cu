#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <ctypecu.h>
#include <assert.h>

static __global__ void g_ctype_test1() {
	printf("ctype_test1\n");
	//fprintf_(stdout, "ctype_test1\n");

	//// ISCTYPE ////
	//extern __forceinline__ __device__ int isctype_(int c, int type);
	bool _0 = isctype('a', 0x02); assert(_0);

	//// ISALNUM, ISALPHA, ISCNTRL, ISDIGIT, ISLOWER, ISGRAPH, ISPRINT, ISPUNCT, ISSPACE, ISUPPER, ISXDIGIT ////
	//extern __forceinline__ __device__ int isalnum_(int c);
	//extern __forceinline__ __device__ int isalpha_(int c);
	//extern __forceinline__ __device__ int iscntrl_(int c);
	//extern __forceinline__ __device__ int isdigit_(int c);
	//extern __forceinline__ __device__ int islower_(int c);
	//extern __forceinline__ __device__ int isgraph_(int c);
	//extern __forceinline__ __device__ int isprint_(int c);
	//extern __forceinline__ __device__ int ispunct_(int c);
	//extern __forceinline__ __device__ int isspace_(int c);
	//extern __forceinline__ __device__ int isupper_(int c);
	//extern __forceinline__ __device__ int isxdigit_(int c);
	bool a0 = isalnum('a'); bool a0n = isalnum('1'); assert(a0 && a0n);
	bool a1 = isalpha('a'); bool a1n = isalpha('A'); assert(a1 && a1n);
	bool a2 = iscntrl('a'); bool a2n = iscntrl('A'); assert(!a2 && !a2n);
	bool a3 = isdigit('a'); bool a3n = isdigit('1'); assert(!a3 && a3n);
	bool a4 = islower('a'); bool a4n = islower('A'); assert(a4 && !a4n);
	bool a5 = isgraph('a'); bool a5n = isgraph('A'); assert(!a5 && !a5n);
	bool a6 = isprint('a'); bool a6n = isprint('A'); assert(a6 && a6n);
	bool a7 = ispunct('a'); bool a7n = ispunct('A'); assert(!a7 && !a7n);
	bool a8 = isspace('a'); bool a8n = isspace(' '); assert(!a8 && a8n);
	bool a9 = isupper('a'); bool a9n = isupper('A'); assert(!a9 && a9n);
	bool aA = isxdigit('a'); bool aAn = isxdigit('A'); assert(aA && aAn);

	//// TOLOWER, TOUPPER, _TOLOWER, _TOUPPER ////
	//extern __forceinline__ __device__ int tolower_(int c);
	//extern __forceinline__ __device__ int toupper_(int c);
	////existing: #define _tolower(c)
	////existing: #define _toupper(c)
	char b0 = tolower('a'); char b0n = tolower('A'); assert(b0 == 'a' && b0n == 'a');
	char b1 = toupper('a'); char b1n = toupper('A'); assert(b1 == 'A' && b1n == 'A');
	char b2 = _toupper('a'); char b2n = _toupper('A'); assert(b2 == 'A' && b2n != 'A');
	char b3 = _tolower('a'); char b3n = _tolower('A'); assert(b3 != 'a' && b3n == 'a');

	//// ISBLANK, ISIDCHAR ////
	//extern __forceinline__ __device__ int isblank_(int c);
	//extern __forceinline__ __device__ int isidchar_(int c);
	bool c0 = isblank(' '); bool c0n = isblank('A'); assert(c0 && !c0n);
	bool c1 = isidchar('a'); bool c1n = isidchar('A'); assert(c1 && c1n);

	//bool d0 = ispoweroftwo(2); bool d0n = ispoweroftwo(3); assert(d0 && !d0n);
	//bool d1 = isalpha2('a'); bool d1n = isalpha2('A'); assert(d1 && d1n);
}
hipError_t ctype_test1() { g_ctype_test1<<<1, 1>>>(); return hipDeviceSynchronize(); }
